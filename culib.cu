#include "hip/hip_runtime.h"
#include <string.h>
#include <stdio.h>

#include "genType.h"
#include "cutil.h"

#define INT '0'
#define FLOAT '1'

#define TO_DEV hipMemcpyHostToDevice
#define TO_HOST hipMemcpyDeviceToHost


char * parseSpec(char *str_spec) {
   char *tok;
   char *spec;
   char i = 0;

   //This is the absolute largest the spec could be
   //assuming it consisted entirely of
   //"int,int,int,int..."
   spec = (char *)malloc((unsigned int)strlen(str_spec) / 4);

   tok = strtok(str_spec, ", ");
   while (tok != NULL) {
      if (!strcmp(tok, "int"))
         spec[i] = INT;
      else
         spec[i] = FLOAT;
      i++;
      tok = strtok(NULL, ", ");
   }
   spec[i] = '\0';

   return spec;
}

__device__ int cudaStrtoi (char *str, char **end) {
   int i = 0;
   char neg = 0;

   while ((*str - 48 < 0 || *str - 48 > 9) && (*str != '-' && *str != '\0' && *str != '.'))
      *str++;

   if (*str == '-') {
      *str++;
      neg = 1;
   }

   while (*str && *str - 48 > 0 && *str - 48 <= 9 && *str != 'e' && *str != '.')
      i = (i << 3) + (i << 1) + ((*str++) - '0');

   *end = str;
   return neg ? -i : i;
}

__device__ float cudaStrtof (char *str, char **end) {
   float f;

   f = cudaStrtoi(str, &str);

   if (*str == '.') {
      *str++;
      char *pos = str;
      f += (f >= 0 ? 1 : -1) * (float)cudaStrtoi(str, &str) / exp10f(str - pos);
   }

   if (*str == 'e') {
      *str++;
      f *= exp10f(cudaStrtoi(str, &str));
   }

   *end = str;
   return f;
}

__device__ int cudaAtoi (char *str) {
   return cudaStrtoi (str, NULL);
}

__device__ float cudaAtof (char *str) {
   return cudaStrtof(str, NULL);
}

__global__ void jsonToObj(char *sObj, char *spec, char *obj) {
   int pos = 0;
   float fres;
   int ires;

   for (int i = 0; spec[i] != '\0'; i++) {
      if (spec[i] == INT) {
         ires = cudaStrtoi(sObj, &sObj);
         memcpy((obj + pos), &ires, sizeof(int));
         pos += sizeof(int);
      }
      else {
         fres = cudaStrtof(sObj, &sObj);
         memcpy((obj + pos), &fres, sizeof(float));
         pos += sizeof(float);
      }
   }
}

GenType parseObjects(char *json, char *spec) {
   char * dev_json;
   char * dev_obj;
   char * dev_spec;
   GenType out;

   CUDA_SAFE_CALL(hipMalloc((void **) &dev_json, strlen(json) + 1));
   CUDA_SAFE_CALL(hipMalloc((void **) &dev_obj, sizeof(GenType)));
   CUDA_SAFE_CALL(hipMalloc((void **) &dev_spec, strlen(spec) + 1));

   CUDA_SAFE_CALL(hipMemcpy(dev_spec, spec, strlen(spec) + 1, TO_DEV));
   CUDA_SAFE_CALL(hipMemcpy(dev_json, json, strlen(json) + 1, TO_DEV));

   jsonToObj<<<1, 1>>>(dev_json, dev_spec, dev_obj);

   CUDA_SAFE_CALL(hipMemcpy((char *) &out, dev_obj, sizeof(GenType), TO_HOST));

   CUDA_SAFE_CALL(hipFree(dev_json));
   CUDA_SAFE_CALL(hipFree(dev_obj));

   return out;
}
