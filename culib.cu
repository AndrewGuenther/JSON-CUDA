#include "hip/hip_runtime.h"
#include <string.h>
#include <stdio.h>

#include "genType.h"
#include "cutil.h"

#define INT '0'
#define FLOAT '1'

#define TO_DEV hipMemcpyHostToDevice
#define TO_HOST hipMemcpyDeviceToHost

int objSize(char *spec) {
   int i = 0, objSize = 0;

   while (spec[i])
      objSize += (spec[i++] == INT ? sizeof(int) : sizeof(float));

   return objSize;
}

char * parseSpec(char *str_spec) {
   char *tok;
   char *spec;
   char i = 0;

   //This is the absolute largest the spec could be
   //assuming it consisted entirely of
   //"int,int,int,int..."
   spec = (char *)malloc((unsigned int)strlen(str_spec) / 4);

   tok = strtok(str_spec, ", ");
   while (tok != NULL) {
      if (!strcmp(tok, "int"))
         spec[i] = INT;
      else
         spec[i] = FLOAT;
      i++;
      tok = strtok(NULL, ", ");
   }
   spec[i] = '\0';

   return spec;
}

__device__ int cudaStrtoi (char *str, char **end) {
   int i = 0;
   char neg = 0;

   while ((*str - 48 < 0 || *str - 48 > 9) && (*str != '-' && *str != '\0' && *str != '.'))
      *str++;

   if (*str == '-') {
      *str++;
      neg = 1;
   }

   while (*str && *str - 48 >= 0 && *str - 48 <= 9 && *str != 'e' && *str != '.')
      i = (i << 3) + (i << 1) + ((*str++) - '0');

   *end = str;
   return neg ? -i : i;
}

__device__ float cudaStrtof (char *str, char **end) {
   float f;
   char neg = 0;

   while ((*str - 48 < 0 || *str - 48 > 9) && (*str != '-' && *str != '\0' && *str != '.'))
      *str++;

   if (*str == '-') {
      *str++;
      neg = 1;
   }

   f = cudaStrtoi(str, &str);

   if (*str == '.') {
      *str++;
      char *pos = str;
      f += (f >= 0 ? 1 : -1) * (float)cudaStrtoi(str, &str) / exp10f(str - pos);
   }

   if (*str == 'e') {
      *str++;
      f *= exp10f(cudaStrtoi(str, &str));
   }

   *end = str;
   return neg ? -f : f;
}

__device__ int cudaAtoi (char *str) {
   return cudaStrtoi (str, NULL);
}

__device__ float cudaAtof (char *str) {
   return cudaStrtof(str, NULL);
}

#define THREADS_PER_BLOCK 512
#define INITIAL_SIZE 1024

__global__ void jsonToObj(char *sObj, char *spec, char *obj, unsigned int * starts, int objSize, int numElements) {
   float fres;
   int ires;
   unsigned int offset = blockIdx.x * blockDim.x + threadIdx.x;

   if (offset >= numElements)
      return;

   obj += offset * objSize;
   sObj += starts[offset];

   for (int i = 0; spec[i] != '\0'; i++) {
      if (spec[i] == INT) {
         ires = cudaStrtoi(sObj, &sObj);
         memcpy(obj, &ires, sizeof(int));
         obj += sizeof(int);
      }
      else {
         fres = cudaStrtof(sObj, &sObj);
         memcpy(obj, &fres, sizeof(float));
         obj += sizeof(float);
      }
   }
}

char *dev_json, *dev_spec;
int size;

char * parseArray(unsigned int *dev_starts, int numElements) {
   char * dev_obj;
   char * out;

   printf("called\n");

   CUDA_SAFE_CALL(hipMalloc((void **) &dev_obj, size * numElements));

   dim3 dimBlock(numElements / THREADS_PER_BLOCK + 1);
   dim3 dimThread(THREADS_PER_BLOCK);

   jsonToObj<<<dimBlock, dimThread>>>(dev_json, dev_spec, dev_obj, dev_starts, size, numElements);

   out = (char *)malloc(size * numElements);

   CUDA_SAFE_CALL(hipMemcpy(out, dev_obj, size * numElements, TO_HOST));

   return out;
}

int depth;

char * findArrays(char *json, char *pos, char **newpos) {
   unsigned int *starts, *dev_starts;
   char *out;
   char **arrs;
   int i = 0;
   char parsing = 1;
   unsigned int numElements = 0;
   unsigned int startsSize = INITIAL_SIZE;

   printf("depth: %d\n", depth);
   depth++;

   starts = (unsigned int *)malloc(sizeof(int) * INITIAL_SIZE);
   arrs = (char **)malloc(5 * sizeof(char **));

   do {
      if (*pos == '[') {
         if (*(pos + 2) != '[') {
            starts[numElements] = pos - json;
            numElements++;
            if (numElements >= startsSize) {
               startsSize += INITIAL_SIZE;
               starts = (unsigned int *)realloc(starts, (sizeof(int) * startsSize));
            }
         }
         else {
            arrs[i] = findArrays(json, pos + 1, &pos);
            i++;
            printf("%c\n", *pos);
            parsing = 0; 
         }
      }
   } while (*++pos != '\0');

   *newpos = pos;
   depth--;

   if (parsing) {
      CUDA_SAFE_CALL(hipMalloc((void **) &dev_starts, numElements * size));
      CUDA_SAFE_CALL(hipMemcpy(dev_starts, starts, numElements * sizeof(int), TO_DEV));

      out = parseArray(dev_starts, numElements);

      CUDA_SAFE_CALL(hipFree(dev_starts));

      return out;
   }
   else
      return (char *)arrs;

}

char * parseObjects(char *json, char *spec, int objSize) {
   char *out;
   char *pos = json;

   size = objSize;
   depth = 0;

   CUDA_SAFE_CALL(hipMalloc((void **) &dev_json, strlen(json) + 1));
   CUDA_SAFE_CALL(hipMalloc((void **) &dev_spec, strlen(spec) + 1)); //Make this constant mem

   CUDA_SAFE_CALL(hipMemcpy(dev_spec, spec, strlen(spec) + 1, TO_DEV));
   CUDA_SAFE_CALL(hipMemcpy(dev_json, json, strlen(json) + 1, TO_DEV));

   out = findArrays(json, pos, &pos);

   CUDA_SAFE_CALL(hipFree(dev_spec));
   CUDA_SAFE_CALL(hipFree(dev_json));

   return out;
}
