#include "hip/hip_runtime.h"
#include <string.h>
#include <stdio.h>

#include "genType.h"
#include "cutil.h"

#define INT '0'
#define FLOAT '1'

#define TO_DEV hipMemcpyHostToDevice
#define TO_HOST hipMemcpyDeviceToHost

int objSize(char *spec) {
   int i = 0, size = 0;

   while (spec[i])
      size += (spec[i++] == INT ? sizeof(int) : sizeof(float));

   return size;
}

char * parseSpec(char *str_spec) {
   char *tok;
   char *spec;
   char i = 0;

   //This is the absolute largest the spec could be
   //assuming it consisted entirely of
   //"int,int,int,int..."
   spec = (char *)malloc((unsigned int)strlen(str_spec) / 4);

   tok = strtok(str_spec, ", ");
   while (tok != NULL) {
      if (!strcmp(tok, "int"))
         spec[i] = INT;
      else
         spec[i] = FLOAT;
      i++;
      tok = strtok(NULL, ", ");
   }
   spec[i] = '\0';

   return spec;
}

__device__ int cudaStrtoi (char *str, char **end) {
   int i = 0;
   char neg = 0;

   while ((*str - 48 < 0 || *str - 48 > 9) && (*str != '-' && *str != '\0' && *str != '.'))
      *str++;

   if (*str == '-') {
      *str++;
      neg = 1;
   }

   while (*str && *str - 48 >= 0 && *str - 48 <= 9 && *str != 'e' && *str != '.')
      i = (i << 3) + (i << 1) + ((*str++) - '0');

   *end = str;
   return neg ? -i : i;
}

__device__ float cudaStrtof (char *str, char **end) {
   float f;

   f = cudaStrtoi(str, &str);

   if (*str == '.') {
      *str++;
      char *pos = str;
      f += (f >= 0 ? 1 : -1) * (float)cudaStrtoi(str, &str) / exp10f(str - pos);
   }

   if (*str == 'e') {
      *str++;
      f *= exp10f(cudaStrtoi(str, &str));
   }

   *end = str;
   return f;
}

__device__ int cudaAtoi (char *str) {
   return cudaStrtoi (str, NULL);
}

__device__ float cudaAtof (char *str) {
   return cudaStrtof(str, NULL);
}

#define THREADS_PER_BLOCK 512
#define INITIAL_SIZE 128

__global__ void jsonToObj(char *sObj, char *spec, char *obj, int * starts, int objSize, int numElements) {
   int pos = 0;
   float fres;
   int ires;
   int offset = blockIdx.x * THREADS_PER_BLOCK + threadIdx.x;

   if (offset > numElements)
      return;

   obj += offset * objSize;
   sObj += starts[offset];

   for (int i = 0; spec[i] != '\0'; i++) {
      if (spec[i] == INT) {
         ires = cudaStrtoi(sObj, &sObj);
         memcpy((obj + pos), &ires, sizeof(int));
         pos += sizeof(int);
      }
      else {
         fres = cudaStrtof(sObj, &sObj);
         memcpy((obj + pos), &fres, sizeof(float));
         pos += sizeof(float);
      }
   }
}

char * parseObjects(char *json, char *spec, int size) {
   char * dev_json;
   char * dev_obj;
   char * dev_spec;
   int * dev_starts;
   char * out;
   unsigned int numElements = 0;
   unsigned int * starts;
   char * pos = json;

   starts = (unsigned int *)malloc(sizeof(int) * INITIAL_SIZE);
   while (*++pos != '\0') {
      if (*pos == '[' && *(pos + 1) != '[') {
         starts[numElements] = pos - json;
         numElements++;
      }
   }

   printf("%d\n", numElements);
   for (int i = 0; i < numElements; i++)
      printf("%d ", starts[i]);
   printf("\n");
   fflush(stdout);

   out = (char *)malloc(size * numElements);

   CUDA_SAFE_CALL(hipMalloc((void **) &dev_starts, numElements * sizeof(int)));
   CUDA_SAFE_CALL(hipMalloc((void **) &dev_json, strlen(json) + 1));
   CUDA_SAFE_CALL(hipMalloc((void **) &dev_obj, size * numElements));
   CUDA_SAFE_CALL(hipMalloc((void **) &dev_spec, strlen(spec) + 1));

   CUDA_SAFE_CALL(hipMemcpy(dev_starts, starts, numElements * sizeof(int), TO_DEV));
   CUDA_SAFE_CALL(hipMemcpy(dev_spec, spec, strlen(spec) + 1, TO_DEV));
   CUDA_SAFE_CALL(hipMemcpy(dev_json, json, strlen(json) + 1, TO_DEV));

   dim3 dimBlock(numElements / THREADS_PER_BLOCK + 1);
   dim3 dimThread(THREADS_PER_BLOCK);
   jsonToObj<<<dimBlock, dimThread>>>(dev_json, dev_spec, dev_obj, dev_starts, size, numElements);

   CUDA_SAFE_CALL(hipMemcpy(out, dev_obj, size * numElements, TO_HOST));

   CUDA_SAFE_CALL(hipFree(dev_json));
   CUDA_SAFE_CALL(hipFree(dev_obj));

   return out;
}
