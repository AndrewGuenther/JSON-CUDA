#include "hip/hip_runtime.h"
#include <string.h>
#include <stdio.h>

#include "genType.h"
#include "cutil.h"

#define INT '0'
#define FLOAT '1'

#define TO_DEV hipMemcpyHostToDevice
#define TO_HOST hipMemcpyDeviceToHost

int objSize(char *spec) {
   int i = 0, objSize = 0;

   while (spec[i])
      objSize += (spec[i++] == INT ? sizeof(int) : sizeof(float));

   return objSize;
}

char * parseSpec(char *str_spec) {
   char *tok;
   char *spec;
   char i = 0;

   //This is the absolute largest the spec could be
   //assuming it consisted entirely of
   //"int,int,int,int..."
   spec = (char *)malloc((unsigned int)strlen(str_spec) / 4);

   tok = strtok(str_spec, ", ");
   while (tok != NULL) {
      if (!strcmp(tok, "int"))
         spec[i] = INT;
      else
         spec[i] = FLOAT;
      i++;
      tok = strtok(NULL, ", ");
   }
   spec[i] = '\0';

   return spec;
}

__device__ int cudaStrtoi (char *str, char **end) {
   int i = 0;
   char neg = 0;

   while ((*str - 48 < 0 || *str - 48 > 9) && (*str != '-' && *str != '\0' && *str != '.'))
      *str++;

   if (*str == '-') {
      *str++;
      neg = 1;
   }

   while (*str && *str - 48 >= 0 && *str - 48 <= 9 && *str != 'e' && *str != '.')
      i = (i << 3) + (i << 1) + ((*str++) - '0');

   *end = str;
   return neg ? -i : i;
}

__device__ float cudaStrtof (char *str, char **end) {
   float f;
   char neg = 0;

   while ((*str - 48 < 0 || *str - 48 > 9) && (*str != '-' && *str != '\0' && *str != '.'))
      *str++;

   if (*str == '-') {
      *str++;
      neg = 1;
   }

   f = cudaStrtoi(str, &str);

   if (*str == '.') {
      *str++;
      char *pos = str;
      f += (f >= 0 ? 1 : -1) * (float)cudaStrtoi(str, &str) / exp10f(str - pos);
   }

   if (*str == 'e') {
      *str++;
      f *= exp10f(cudaStrtoi(str, &str));
   }

   *end = str;
   return neg ? -f : f;
}

__device__ int cudaAtoi (char *str) {
   return cudaStrtoi (str, NULL);
}

__device__ float cudaAtof (char *str) {
   return cudaStrtof(str, NULL);
}

#define THREADS_PER_BLOCK 512
#define INITIAL_SIZE 1024
#define ARRS_SIZE 1024

__global__ void jsonToObj(char *sObj, char *spec, char *obj, unsigned int * starts, int objSize, int numElements) {
   float fres;
   int ires;
   unsigned int offset = blockIdx.x * blockDim.x + threadIdx.x;

   if (offset >= numElements)
      return;

   obj += offset * objSize;
   sObj += starts[offset];

   for (int i = 0; spec[i] != '\0'; i++) {
      if (spec[i] == INT) {
         ires = cudaStrtoi(sObj, &sObj);
         memcpy(obj, &ires, sizeof(int));
         obj += sizeof(int);
      }
      else {
         fres = cudaStrtof(sObj, &sObj);
         memcpy(obj, &fres, sizeof(float));
         obj += sizeof(float);
      }
   }
}

char *dev_json, *dev_spec;
int size;

char * parseArray(unsigned int *dev_starts, int numElements) {
   char * dev_obj;
   char * out;

   CUDA_SAFE_CALL(hipMalloc((void **) &dev_obj, size * numElements));

   dim3 dimBlock(numElements / THREADS_PER_BLOCK + 1);
   dim3 dimThread(THREADS_PER_BLOCK);

   jsonToObj<<<dimBlock, dimThread>>>(dev_json, dev_spec, dev_obj, dev_starts, size, numElements);

   out = (char *)malloc(size * numElements);

   CUDA_SAFE_CALL(hipMemcpy(out, dev_obj, size * numElements, TO_HOST));

   return out;
}

int depth;

char * setupArray(char *json, char *pos, char **newpos) {
   unsigned int *starts, *dev_starts;
   char *out;
   int balance = 0;
   unsigned int numElements = 0;
   unsigned int startsSize = INITIAL_SIZE;  
//   GenType * debug_out;
   
   starts = (unsigned int *)malloc(sizeof(int) * INITIAL_SIZE);

//   printf("setupArray: ");
   do {
      if (*pos == '[') {
         balance++;
         starts[numElements] = pos - json;
         numElements++;
         if (numElements >= startsSize) {
            startsSize += INITIAL_SIZE;
            starts = (unsigned int *)realloc(starts, (sizeof(int) * startsSize));
         }
      }
      else if (*pos == ']')
         balance--;
//      printf("%c", *pos);
      fflush(stdout);
   } while (*++pos != '\0' && balance >= 0);

//   printf("\n");
   *newpos = pos - 1;

   CUDA_SAFE_CALL(hipMalloc((void **) &dev_starts, numElements * size));
   CUDA_SAFE_CALL(hipMemcpy(dev_starts, starts, numElements * sizeof(int), TO_DEV));

   out = parseArray(dev_starts, numElements);
//   debug_out = (GenType *)out;
//   for (int i = 0; i < numElements; i++)
//      printf("%d, %.2lf, %d, %d, %.2lf\n", debug_out[i].a, debug_out[i].b, debug_out[i].c, debug_out[i].d, debug_out[i].e);

   CUDA_SAFE_CALL(hipFree(dev_starts));

   return out;
}

char * findArrays(char *json, char *pos, char **newpos) {
   char *out;
   char **arrs;
   unsigned int arrs_size = ARRS_SIZE;
   int i = 0, balance = 0;
   char parsed = 0;

   arrs = (char **)malloc(arrs_size * sizeof(char **));

   pos++;
//   printf("Find arrays %d: ", depth);
   if (*pos == '[') {
      if(*(pos + 1) != '[') {
  
//         printf("%c", *pos);
//         printf("\n");
         out = setupArray(json, pos, &pos);
//         printf("%x\n", out);
//         printf("%c", *pos);
         parsed = 1;
      }
      else {
         do {
            if (*pos == '[') {
               balance++;
//               printf("\n");
               depth++;
//               printf("down\n");
               arrs[i] = findArrays(json, pos, &pos);
               i++;
               if (i >= arrs_size) {
                  printf("resizing\n");
                  arrs_size += ARRS_SIZE;
                  arrs = (char **)realloc(arrs, (arrs_size * sizeof(char **)));
               }
//               printf("up\n");
               depth--;
            }
            if (*pos == ']')
               balance--;
//            printf("%c", *pos);
            fflush(stdout);
         } while (*++pos != '\0' && balance >= 0);
      }
   }

//   printf("\n");
   *newpos = pos;


   if (parsed)
      return out;
   else {
//      printf("%x: ", arrs);
//      for (int j = 0; j < i; j++)
//         printf("%x, ", arrs[j]);
//      printf("\n");

      return (char *)arrs;
   }
}

char * parseObjects(char *json, char *spec, int objSize) {
   char *out;
   char *pos = json;

   size = objSize;
   depth = 0;

   CUDA_SAFE_CALL(hipMalloc((void **) &dev_json, strlen(json) + 1));
   CUDA_SAFE_CALL(hipMalloc((void **) &dev_spec, strlen(spec) + 1)); //Make this constant mem

   CUDA_SAFE_CALL(hipMemcpy(dev_spec, spec, strlen(spec) + 1, TO_DEV));
   CUDA_SAFE_CALL(hipMemcpy(dev_json, json, strlen(json) + 1, TO_DEV));

   out = findArrays(json, pos, &pos);

   CUDA_SAFE_CALL(hipFree(dev_spec));
   CUDA_SAFE_CALL(hipFree(dev_json));

   return out;
}
