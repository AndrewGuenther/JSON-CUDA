#include "hip/hip_runtime.h"
#include <string.h>
#include <stdio.h>

#include "genType.h"

#define INT '0'
#define FLOAT '1'

char * parseSpec(char *str_spec) {
   char *tok;
   char *spec;
   char i = 0;

   //This is the absolute largest the spec could be
   //assuming it consisted entirely of
   //"int,int,int,int..."
   spec = (char *)malloc((unsigned int)strlen(str_spec) / 4);

   printf("%s\n", str_spec);
   fflush(stdout);

   tok = strtok(str_spec, ", ");
   while (tok != NULL) {
      if (!strcmp(tok, "int"))
         spec[i] = INT;
      else
         spec[i] = FLOAT;
      i++;
      tok = strtok(NULL, ", ");
   }

   return spec;
}

void parseObjects(char *json, char *spec) {
   
}

__global__ void jsonToObj(char *obj, char *spec) {

}

__device__ int cudaStrtoi (char *str, char **end) {
   int i = 0;
   char neg = 0;

   if (*str == '-') {
      *str++;
      neg = 1;
   }

   while (*str && *str != 'e' && *str != '.')
      i = (i << 3) + (i << 1) + ((*str++) - '0');

   *end = str;
   return neg ? -i : i;
}

__device__ int cudaAtoi (char *str) {
  return cudaStrtoi (str, &str);
}


__device__ float cudaAtof (char *str) {
   float f;

   f = cudaStrtoi(str, &str);

   if (*str == '.') {
      *str++;
      char *pos = str;
      f += (float)cudaStrtoi(str, &str) / exp10f(str - pos);
   }

   if (*str == 'e') {
      *str++;
      f *= exp10f(cudaStrtoi(str, &str));
   }

   return f;
}
